#include "hip/hip_runtime.h"
#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "utils.h"
//PASSES MAKE CHECK

__global__ void flipKernel(float*arr, float *out, int w){
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	// int x = blockIdx.x*16+threadIdx.x;
	// int y = blockIdx.y*16+threadIdx.y;
	out[y*w+w-x-1]=arr[y*w+x];
}

void flip(float *arr, float *out, int w) {
	dim3 dim_blocks(w/16,w/16);
	dim3 dim_threads(16,16,1);
	flipKernel<<<dim_blocks, dim_threads>>>(arr, out, w);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("");

}

__global__ void transposeKernel(float*arr, float *out, int w){
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	//int x = blockIdx.x*16+threadIdx.x;
	//int y = blockIdx.y*16+threadIdx.y;
	//out[y*w+x]=arr[y+x*w];
	out[y+x*w]=arr[y*w+x];
}

void transpose(float *arr, float *out, int w) {
	dim3 dim_blocks(w/16,w/16);
	dim3 dim_threads(16,16,1);
	transposeKernel<<<dim_blocks, dim_threads>>>(arr, out, w);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("");

}



__global__ void reductionKernel(float* A, int len, int stride) {
	int  index = blockIdx.x*blockDim.x+threadIdx.x;
	if (index+stride<len){
		index+=blockIdx.y*len;
		A[index] += A[index+stride];
	}
}



void reductionGPU(float* A, int len) {

	int threads_per_block = 512; 
	int blocks_per_grid = (len/1024);
	if (blocks_per_grid==0){
		blocks_per_grid=1;
	}
	int stride = len/2;

	while (stride>=1) { 
		dim3 dim_blocks(blocks_per_grid,1);
		dim3 dim_threads(threads_per_block,1,1);
		reductionKernel<<<dim_blocks, dim_threads>>>(A,len,stride);
		hipDeviceSynchronize();
		CUT_CHECK_ERROR("");

		stride /=2;

		if (blocks_per_grid!=1){
			blocks_per_grid /=2;
		}
		else if(threads_per_block != 1){
			threads_per_block /= 2;
		}
	}

}



__global__ void calcdistKernel(float* out, float* img, int i_w, int i_h, float* t, int t_w, int x, int y) {
	int index0 = 	blockIdx.x*16 + threadIdx.x
					+ (blockIdx.y)*t_w
					+ t_w*t_w*threadIdx.z;	
	int index1 = 	blockIdx.x*16 + threadIdx.x + x
					+ (blockIdx.y+y)*i_w;
	//int index2 = 	index0 + t_w*t_w*threadIDx.z;	
	float diff = img[index1]-t[index0];
	out[index0] = diff*diff;
}


float calc_min_dist(float *img, int i_w, int i_h, float *t, int t_w) {

	float min_dist = FLT_MAX;
	float* dist = (float*)malloc(sizeof(float));
	int x = i_w-t_w;
	int y = i_h-t_w;

	float* t2,* t3,* t4;
	
	int t_len = t_w*t_w;
	int t_len2 = t_len*sizeof(float);
	//int len = 8*t_w*t_w*sizeof(float);
	CUDA_SAFE_CALL(hipMalloc(&t2, t_len2));
	CUDA_SAFE_CALL(hipMalloc(&t3, t_len2));
	float* out;
	CUDA_SAFE_CALL(hipMalloc(&out, t_len2));

	CUDA_SAFE_CALL(hipMemcpy(t2, t,t_len2,hipMemcpyDeviceToDevice)); 	
	// //t_len *=4;
	// flip(t, t2+t_len2, t_w);
	// //flip(t2+t_len2, t2, t_w);
	// transpose(t2+t_len2, t2+t_len2*2, t_w);
	// flip(t2+t_len2*2, t2+t_len2*3, t_w);
	// transpose(t2+t_len2*3, t2+t_len2*4, t_w);
	// flip(t2+t_len2*4, t2+t_len2*5, t_w);
	// transpose(t2+t_len2*5, t2+t_len2*6, t_w);
	// flip(t2+t_len2*6, t2+t_len2*7, t_w);
	// //t_len/=4;

	for (int k = 1; k<9;k++){

		if (k%2==0){
			flip(t2, t3, t_w);
			t4=t2;
			t2=t3;
			t3=t4;
		}else if(k!=1){
			transpose(t2, t3, t_w);
			t4=t2;
			t2=t3;
			t3=t4;
		}


		for (int i=0; i<=x;i++){
			for (int j=0; j<=y;j++){
				//printf("k:%d/n",k);
				dim3 dim_blocks(t_w/16,t_w);
				dim3 dim_threads(16, 1, 1);
				calcdistKernel<<<dim_blocks, dim_threads>>>(out, img, i_w, i_h, t2, t_w, i, j);
				hipDeviceSynchronize();
				CUT_CHECK_ERROR("");

				reductionGPU(out, t_len);

				CUDA_SAFE_CALL(hipMemcpy(dist, out,sizeof(float),hipMemcpyDeviceToHost)); 
				if(*dist<min_dist) min_dist=*dist;
				//int offset=t_len;
				// CUDA_SAFE_CALL(hipMemcpy(dist, out+offset,sizeof(float),hipMemcpyDeviceToHost)); 
				// if(*dist<min_dist) min_dist=*dist;
				// offset+=t_len;
				// CUDA_SAFE_CALL(hipMemcpy(dist, out+offset,sizeof(float),hipMemcpyDeviceToHost)); 
				// if(*dist<min_dist) min_dist=*dist;
				// offset+=t_len;
				// CUDA_SAFE_CALL(hipMemcpy(dist, out+offset,sizeof(float),hipMemcpyDeviceToHost)); 
				// if(*dist<min_dist) min_dist=*dist;
				// offset+=t_len;
				// CUDA_SAFE_CALL(hipMemcpy(dist, out+offset,sizeof(float),hipMemcpyDeviceToHost)); 
				// if(*dist<min_dist) min_dist=*dist;
				// offset+=t_len;
				// CUDA_SAFE_CALL(hipMemcpy(dist, out+offset,sizeof(float),hipMemcpyDeviceToHost)); 
				// if(*dist<min_dist) min_dist=*dist;
				// offset+=t_len;
				// CUDA_SAFE_CALL(hipMemcpy(dist, out+offset,sizeof(float),hipMemcpyDeviceToHost)); 
				// if(*dist<min_dist) min_dist=*dist;
				// offset+=t_len;
				// CUDA_SAFE_CALL(hipMemcpy(dist, out+offset,sizeof(float),hipMemcpyDeviceToHost)); 
				// if(*dist<min_dist) min_dist=*dist;
			}
		}
	}

	CUDA_SAFE_CALL(hipFree(t2));
	CUDA_SAFE_CALL(hipFree(out));

	return min_dist;
}
